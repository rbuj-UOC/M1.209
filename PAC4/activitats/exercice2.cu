#include "hip/hip_runtime.h"
%%cuda
#include <iostream>

#define N 10

using namespace std;

__global__ void multipicationKernel(const int size,
                                    const double *src_matrix_1,
                                    const double *src_matrix_2,
                                    double *dst_matrix)
{
  // express the collection of blocks, and the collection of threads
  // within a block, as a 2-D array
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  // Typical problems are not friendly multiples of blockDim.x
  // Avoid accessing beyond the end of the arrays
  if (row < size && col < size)
  {
    double sum = 0;
    for (int k = 0; k < size; k++)
      sum += src_matrix_1[row * size + k] * src_matrix_2[k * size + col];
    dst_matrix[row * size + col] = sum;
  }
}

void print_matrix(double m[N][N], string name) {
  std::cout << name << endl;
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      std::cout << m[i][j] << ", ";
    }
    std::cout << endl;
  }
}

int main()
{
  double a[N][N], b[N][N], c[N][N];
  double *d_a, *d_b, *d_c;
  size_t size = N * N * sizeof(double);
  struct timespec t0, t1;

  // alloc space for device copies of a, b, and c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // initialize matrices
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      a[i][j] = i;
      b[i][j] = j;
    }
  }

  // copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  dim3 dimGrid(1, 1); // grid = 1 x 1 block
  dim3 dimBlock(N, N); // block = N x N threads

  multipicationKernel<<<dimGrid, dimBlock>>>(N, d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  print_matrix(a, "A");
  print_matrix(b, "B");
  print_matrix(c, "C");

  // clean up
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}