#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>

__global__ void kernel(){
  // express the collection of blocks, and the collection of threads within a block, as a 1-D array
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bdim = blockDim.x;
  int idx = bid * bdim + tid;
  printf("My Id is %d, I am the thread %d of %d in the block %d\n",
         idx, tid, bdim, bid);
}

int main(){
  int bnum = 1;
  int tnum = 9;
  printf("Username: capa08\n");
  printf("Blocks: %d\n", bnum);
  printf("Threads per block: %d\n", tnum);
  kernel<<<bnum,tnum>>>();
  hipDeviceSynchronize();
  return 0;
}