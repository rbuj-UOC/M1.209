#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <iostream>
#include <fstream>

//#define INPUT_FILE "test_image.ppm" 
//#define OUTPUT_FILE "output_image.ppm" 

//#define INPUT_FILE "/content/sample_data/test_image.ppm" 
//#define OUTPUT_FILE "/content/sample_data/output_image.ppm" 

#define INPUT_FILE "/content/drive/MyDrive/test_image.ppm" 
#define OUTPUT_FILE "/content/drive/MyDrive/output_image.ppm" 

using namespace std; 

void savePPM(const char *filename, 
             unsigned char *data, 
             int width, 
             int height) { 
    ofstream file(filename, ios::binary); 
    file << "P5\n" << width << " " << height << "\n255\n";

    // Write grayscale data 
    file.write(reinterpret_cast<char *>(data), width * height); 
    file.close(); 
} 

bool loadPPM(const char *filename, 
             unsigned char *data, 
             int width, 
             int height) { 
    ifstream file(filename, ios::binary); 
    if (!file) { 
      cerr << "couldn't find file: " << filename << endl; 
      return false; 
    } 

    string header; 
    file >> header >> width >> height;  // P6, width, height 
    int maxVal; 
    file >> maxVal; 
    file.ignore(); // skip newline 

    file.read(reinterpret_cast<char *>(data), width * height * 3); 
    return true; 
} 

__global__ void kernel(unsigned char *d_A, unsigned char *d_B, int width, int height){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int p = y * width + x; 
  int idx = p * 3; 
    
  // Typical problems are not friendly multiples of blockDim.x 
  // Avoid accessing beyond the end of the arrays 
  if (x < width && y < height) { 
      float gray = 0.3f * d_A[idx] + 
                   0.59f * d_A[idx + 1] + 
                   0.11f * d_A[idx + 2]; 
      d_B[p] = gray;
/*
      printf("x: %d, y: %d %u\n", x, y, (unsigned int)gray);
*/
  } 
}

int main(){
  const int width = 1920, height = 1080; 
  const int n_pixels = width * height;
  const size_t bytes_A = n_pixels * sizeof(unsigned char) * 3; 
  const size_t bytes_B = n_pixels * sizeof(unsigned char); 
  int n = 10;

  // Allocate memory for arrays A, B on host 
  unsigned char *A, *B;
  A = (unsigned char*)malloc(bytes_A);
  B = (unsigned char*)malloc(bytes_B);

  // Allocate memory for arrays d_A, d_B on device 
  unsigned char *d_A, *d_B; 
  hipMalloc(&d_A, bytes_A); 
  hipMalloc(&d_B, bytes_B);

  if (!loadPPM(INPUT_FILE, A, width, height)) { 
    // Cleanup 
    free(A); 
    free(B); 
    hipFree(d_A); 
    hipFree(d_B); 
    // Raise SIGABRT 
    abort(); 
  } 

  // Copy data from host array A to device array d_A
  hipMemcpy(d_A, A, bytes_A, hipMemcpyHostToDevice); 

  dim3 dimGrid(192, 108);
  dim3 dimBlock(n, n);
  kernel<<<dimGrid,dimBlock>>>(d_A, d_B, width, height);

  // Copy data from device array d_B to host array B 
  hipMemcpy(B, d_B, bytes_B, hipMemcpyDeviceToHost); 

  savePPM(OUTPUT_FILE, B, width, height); 

  // Cleanup 
  free(A); 
  free(B); 
  hipFree(d_A); 
  hipFree(d_B); 

  return 0;
}