#include "hip/hip_runtime.h"
%%cuda
#include <iostream>
#include <random>
#include <ctime>
#include <iomanip>

#define N 1000
#define DIVISORS 16

using namespace std;

__global__ void multipicationKernel(const int size,
                                    const double *src_matrix_1,
                                    const double *src_matrix_2,
                                    double *dst_matrix)
{
  // express the collection of blocks, and the collection of threads within a
  // block, as a 2-D array
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  // Typical problems are not friendly multiples of blockDim.x
  // Avoid accessing beyond the end of the arrays
  if (row < size && col < size)
  {
    double sum = 0;
    for (int k = 0; k < size; k++)
      sum += src_matrix_1[row * size + k] * src_matrix_2[k * size + col];
    dst_matrix[row * size + col] = sum;
  }
}

int main()
{
  double *a, *b, *c;
  double *d_a, *d_b, *d_c;
  size_t size = N * N * sizeof(double);
  struct timespec t0, t1;
  int divisors[DIVISORS] = {1, 2, 4, 5, 8, 10, 20, 25,
                            40, 50, 100, 125, 200, 250, 500, 1000};

  // alloc space for host copies of a, b, and c
  a = (double *) malloc(size);
  b = (double *) malloc(size);
  c = (double *) malloc(size);

  // alloc space for device copies of a, b, and c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // initialize matrices
  double lower_bound = 0;
  double upper_bound = 10;
  uniform_real_distribution<double> unif(lower_bound, upper_bound);
  default_random_engine re;
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      int idx = (i*N) + j;
      a[idx] = unif(re);
      b[idx] = unif(re);
    }
  }

  // copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  cout << "# blocks / threads, execution time" << endl;
  for (int i=0; i<DIVISORS; i++) {
    dim3 dimGrid(divisors[i], divisors[i]);
    dim3 dimBlock(divisors[DIVISORS-i-1], divisors[DIVISORS-i-1]);

    timespec_get(&t0, TIME_UTC);
    multipicationKernel<<<dimGrid, dimBlock>>>(N, d_a, d_b, d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    timespec_get(&t1, TIME_UTC);
    double diff = (double)(t1.tv_sec - t0.tv_sec) + ((double)(t1.tv_nsec - t0.tv_nsec)/1000000000);
    cout << divisors[i] << "b / " << N/divisors[i] << "t, "
         << fixed << setprecision(9) << diff << endl;
  }

  // clean up
  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}